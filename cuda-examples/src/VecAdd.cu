#include "hip/hip_runtime.h"
#include <stdio.h>
#include <timer.h>

#include <iostream>
#include <kernels.cuh>

#define SIZE_OF_ARRAYS 100000000
#define NUM_STREAMS 32


__global__ void add_kernel(float* A, float* B, float* C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N ) {
        for (int i = 0; i < N; i++) {
            C[i] = A[i] + B[i];
        }
    }
}

void launch_vector_add_kerenel_with_streams() {
    int N = SIZE_OF_ARRAYS;
    float* A = (float*)malloc(N * sizeof(float));
    float* B = (float*)malloc(N * sizeof(float));
    float* C = (float*)malloc(N * sizeof(float));
    for (int i = 0; i < N; i++) {
        A[i] = 1.2f;
        B[i] = 2.1f;
    }
    // Allocate memory on device
    float* A_d;
    float* B_d;
    float* C_d;
    hipMalloc((void**)&A_d, N * sizeof(float));
    hipMalloc((void**)&B_d, N * sizeof(float));
    hipMalloc((void**)&C_d, N * sizeof(float));

    // create streams
    hipStream_t streams[NUM_STREAMS];
    for (unsigned int i = 0; i < NUM_STREAMS; i++) {
        hipStreamCreate(&streams[i]);
    }

    // segment the input data and assign to different streams
    unsigned int numSegments = NUM_STREAMS;
    unsigned int segmentSize = ceil( N / ((float) numSegments));
    for (int s = 0; s < numSegments; s++) {
        int start = s * segmentSize;
        int end = (start + segmentSize < N) ? start + segmentSize - 1 : N - 1;
        int segLength = end - start;
        // copy data from host to device in segments 
        hipMemcpyAsync(&A_d[start], &A[start], segLength * sizeof(float), hipMemcpyHostToDevice, streams[s]);
        hipMemcpyAsync(&B_d[start], &B[start], segLength * sizeof(float), hipMemcpyHostToDevice, streams[s]);

        // launch the kernel
        unsigned int numThreadsPerBlock = 512;
        unsigned int numBlocks = ceil(N / (float) numThreadsPerBlock);
        add_kernel<<<numBlocks, numThreadsPerBlock, 0, streams[s]>>>(&A_d[start], &B_d[start], &C_d[start], segmentSize);

        // copy data from device to host in segments
        hipMemcpyAsync(&C[start], &C_d[start], segLength * sizeof(float), hipMemcpyDeviceToHost, streams[s]);
    }

    // wait for all streams to be completed.
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) 
        printf("Error: %s\n", hipGetErrorString(err));

    std::cout << "C[10] value is: " << C[10] << std::endl;
    std::cout << "C[92159999] value is: " << C[92159999] << std::endl;
    std::cout << "C[92160000] value is: " << C[92160000] << std::endl;

    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}